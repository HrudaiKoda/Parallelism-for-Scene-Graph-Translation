#include "hip/hip_runtime.h"
/*
	CS 6023 Assignment 3. 
	Do not make any changes to the boiler plate code or the other files in the folder.
	Use hipFree to deallocate any memory not in usage.
	Optimize as much as possible.
 */

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <stack>

#include<iostream>
class Node {
public:
    int val;
    Node* next;

    Node(int v) : val(v), next(nullptr) {}
};




void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input. 
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;
	

	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ; 
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ; 
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}


void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL; 
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}
	
	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}

int call(int ind,int* hOffset,int* hCsr,std::vector<int>&subtree_nodes_count)
{
  int res = 1;
  for(int i = 0 ; i < *(hOffset+ind+1) - *(hOffset+ind) ;i++)
  {
    res+=call(*(hCsr+*(hOffset+ind)+i),hOffset,hCsr,subtree_nodes_count);
  }
  subtree_nodes_count[ind] = res;
  return res;	 
}



std::vector<int> flatten(const std::vector<std::vector<int>>& vec) {
    std::vector<int> flattened;
    for (const auto& inner_vec : vec) {
        flattened.insert(flattened.end(), inner_vec.begin(), inner_vec.end());
    }
    return flattened;
}

__global__ void scenceGen(int V,int frameSizeX,int frameSizeY,int* gpuScene,int* gpuhOpacity,int* gpuhFrameSizeX,int* gpuhFrameSizeY,int* gpuX,int* gpuY,int** mesh_matrix)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(id < frameSizeX*frameSizeY)
	{
		int row = id/frameSizeY;
		int col = id % frameSizeY;
    int data = 0;
		int localOpacity = 0;
		for(int iter = 0; iter < V;iter++)
		{
			if(*(gpuX+iter) <= row && *(gpuY+iter) <= col && (*(gpuX+iter) + *(gpuhFrameSizeX+iter)) > row && (*(gpuY+iter) + *(gpuhFrameSizeY+iter)) > col )
			{
        if(*(gpuhOpacity+iter) > localOpacity )
        {
          localOpacity = *(gpuhOpacity+iter);
          data = *(*(mesh_matrix+iter) + (row - *(gpuX+iter))* (*(gpuhFrameSizeY+iter)) + (col-*(gpuY+iter)));
        }
			
			}
		}
		*(gpuScene+id) = data;
		
	}
}

__global__ void translateMeshes(int V,int T,int* gpuX,int*gpuY,int* gpu_translations,int*gpu_nodeCount,int*gpuMap,int* gpuPreOrder)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < T)
	{
		int root,dir,step,node;
		root = *(gpu_translations + id*3);
		dir = *(gpu_translations + id*3 +1);
		step = *(gpu_translations + id*3 + 2);
		int ind = *(gpuMap+root);
		for(int iter = 0; iter < *(gpu_nodeCount+root); iter++)
		{
			
			node = *(gpuPreOrder+ind+iter);
			if(dir == 0)
			{
				atomicAdd(gpuX + node, -1*step);
			}
			else if(dir == 1)
			{
				atomicAdd(gpuX + node, step);
			}
			else if(dir == 2)
			{
				atomicAdd(gpuY + node, -1*step);
			}
			else
			{
				atomicAdd(gpuY + node, step);
				
			}
		}
	}
}
    

int main (int argc, char **argv) {
	
	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ; 

	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;
	
	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;  
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now ();

	// Code begins here.
	// Do not change anything above this comment.
	
    int* gpuScene;
  	hipMalloc(&gpuScene,  frameSizeX * frameSizeY * sizeof(int));

	int blocks1 = ceil( frameSizeX * frameSizeY / 1024.0);


	int* gpuhOpacity;
    hipMalloc(&gpuhOpacity,  V* sizeof(int));

	int *gpuhFrameSizeX;
	int *gpuhFrameSizeY;
	hipMalloc(&gpuhFrameSizeX,  V* sizeof(int));
	hipMalloc(&gpuhFrameSizeY,  V* sizeof(int));

	hipMemcpy(gpuhOpacity, hOpacity,V * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuhFrameSizeX, hFrameSizeX,V * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuhFrameSizeY, hFrameSizeY,V * sizeof(int), hipMemcpyHostToDevice);

	int** mesh_matrix;
	hipMalloc(&mesh_matrix, V * sizeof(int*)); 
	for (int i = 0; i < V; ++i) {
		int* temp_mesh_mat;
		hipMalloc(&temp_mesh_mat, *(hFrameSizeX+i) * *(hFrameSizeY+i) * sizeof(int)); 
		hipMemcpy(temp_mesh_mat, hMesh[i], *(hFrameSizeX+i) * *(hFrameSizeY+i)  * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(mesh_matrix + i, &temp_mesh_mat,sizeof(int*), hipMemcpyHostToDevice); 
	}
	
    int* gpuX;
    int* gpuY;

    hipMalloc(&gpuX, V * sizeof(int));
    hipMalloc(&gpuY, V * sizeof(int));

    hipMemcpy(gpuX, hGlobalCoordinatesX, V * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuY, hGlobalCoordinatesY, V * sizeof(int), hipMemcpyHostToDevice);


	std::vector<int> flattened = flatten(translations);
    int* gpu_translations;
    hipMalloc(&gpu_translations, flattened.size() * sizeof(int));

    hipMemcpy(gpu_translations, flattened.data(), flattened.size() * sizeof(int), hipMemcpyHostToDevice);

	int blocks = ceil(numTranslations / 1024.0);
	
  
	int* preOrder = (int*)malloc(sizeof(int)*V);
	int* map = (int*)malloc(sizeof(int)*V);

	int* computationOffset = (int*)malloc(sizeof(int)*V);

	int counter = 0;
	std::stack<int> s;
	int top;
	
	s.push(0);
	while(!s.empty())
	{
		top = s.top();
		s.pop();
		*(preOrder+counter) = top;
		counter++;
		for(int i = 0 ; i < *(hOffset+top+1) - *(hOffset+top) ;i++)
		{
			s.push(*(hCsr+*(hOffset+top)+i));
		}
	}

	for(int iter = 0; iter < V; iter++)
	{
		*(map+*(preOrder+iter)) = iter;
	}

  
	int n = V;  // Number of vertices
   	std::vector<int> subtree_nodes_count(n,0);
   	call(0,hOffset,hCsr,subtree_nodes_count);
    

	int* gpuMap;
	hipMalloc(&gpuMap,V*sizeof(int));
	hipMemcpy(gpuMap,map,V*sizeof(int),hipMemcpyHostToDevice);

	int* gpuPreOrder;
	hipMalloc(&gpuPreOrder,V*sizeof(int));
	hipMemcpy(gpuPreOrder,preOrder,V*sizeof(int),hipMemcpyHostToDevice);


    int* gpu_nodeCount;
    hipMalloc(&gpu_nodeCount, V * sizeof(int));

    hipMemcpy(gpu_nodeCount, subtree_nodes_count.data(), V * sizeof(int), hipMemcpyHostToDevice);

	translateMeshes<<<blocks,1024>>>(V,numTranslations,gpuX,gpuY,gpu_translations,gpu_nodeCount,gpuMap,gpuPreOrder);
	hipFree(gpu_translations);
	hipFree(gpuPreOrder);
	hipFree(gpuMap);
	free(computationOffset);
	computationOffset = NULL;
	free(map);
	map = NULL;
  	scenceGen<<<blocks1,1024>>>(V,frameSizeX,frameSizeY,gpuScene,gpuhOpacity,gpuhFrameSizeX,gpuhFrameSizeY,gpuX,gpuY,mesh_matrix);

	hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(error));
  }
	hipDeviceSynchronize();
	hipMemcpy(hFinalPng,gpuScene,frameSizeX*frameSizeY*sizeof(int),hipMemcpyDeviceToHost);
	// Do not change anything below this comment.
	// Code ends here.

	auto end  = std::chrono::high_resolution_clock::now () ;

	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken) ;
	// Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;	

}
